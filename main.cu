#include "hip/hip_runtime.h"
#include "book.h"

/// Simple Cuda addition function
__global__ void add(int num_one, int num_b, int *sum_result_ptr) {
    *sum_result_ptr = num_one + num_b;
}



/// Query the cuda device(s) for their info
void get_device_info(int &count);


/// Show the results of the cuda addition
void display_add_result(int &host_value, int *&cuda_gpu_ptr);

int main() {
    int host_value;  // "host" means the regular cpu/memory
    int *cuda_gpu_ptr; // "device" mean the CUDA code running on the GPU/MEM
    int count;
    get_device_info(count);
    display_add_result(host_value, cuda_gpu_ptr);


    return 0;
}

void display_add_result(int &host_value, int *&cuda_gpu_ptr) {// allocate memory on the cuda device
    HANDLE_ERROR(hipMalloc((void **) &cuda_gpu_ptr, sizeof(int)));

    // perform calculation with CUDA on GPU
    add<<<1, 1>>>(2, 7, cuda_gpu_ptr);


    // Copy result from CUDA memory back to host memory
    HANDLE_ERROR(hipMemcpy(&host_value, cuda_gpu_ptr, sizeof(int), hipMemcpyDeviceToHost));
    // end utility macro

    printf("3+7 = %d\n", host_value);
    hipFree(cuda_gpu_ptr);
}

void get_device_info(int &count) {
    hipDeviceProp_t device_specs;

    HANDLE_ERROR(hipGetDeviceCount(&count));

    for (int i = 0; i < count; i++) {
        HANDLE_ERROR(hipGetDeviceProperties(&device_specs, i));
    }
}
