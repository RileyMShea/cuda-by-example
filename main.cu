#include "hip/hip_runtime.h"
#include "book.h"
#include "chp3.h"


int main() {
    int host_value;  // "host" means the regular cpu/memory
    int *cuda_gpu_ptr; // "device" mean the CUDA code running on the GPU/MEM
    int count;
    hipDeviceProp_t device_specs;
    get_device_info(count, device_specs);
    display_add_result(host_value, cuda_gpu_ptr);

    return 0;
}

/// Simple addition function
__global__ void add(int num_one, int num_b, int *sum_result_ptr) {
    *sum_result_ptr = num_one + num_b;
}

void display_add_result(int &host_value, int *&cuda_gpu_ptr) {// allocate memory on the cuda device
    HANDLE_ERROR(hipMalloc((void **) &cuda_gpu_ptr, sizeof(int)));

    // perform calculation with CUDA on GPU
    add<<<1, 1>>>(2, 7, cuda_gpu_ptr);


    // Copy result from CUDA memory back to host memory
    HANDLE_ERROR(hipMemcpy(&host_value, cuda_gpu_ptr, sizeof(int), hipMemcpyDeviceToHost));
    // end utility macro

    printf("3+7 = %d\n", host_value);
    hipFree(cuda_gpu_ptr);
}

// TODO: make this a pure function
void get_device_info(int &cuda_device_count, hipDeviceProp_t &device_specs) {

    HANDLE_ERROR(hipGetDeviceCount(&cuda_device_count));

    for (int cuda_device_i = 0; cuda_device_i < cuda_device_count; cuda_device_i++) {

        // Get the device specs for a device
        HANDLE_ERROR(hipGetDeviceProperties(&device_specs, cuda_device_i));

        printf("Device Name: %s\n", device_specs.name);
        printf("Compute capability: %d.%d\n",prop.major, prop.minor)
    }
}
