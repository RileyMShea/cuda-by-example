#include "hip/hip_runtime.h"
#include "book.h"

/// Simple Cuda addition function
__global__ void add(int numberA, int numberB, int *sum_result_ptr) {
    *sum_result_ptr = numberA + numberB;
}

int main() {
    int host_value;  // "host" means the regular cpu/memory
    int *cuda_gpu_ptr; // "device" mean the CUDA code running on the GPU/MEM
    int count;
    hipDeviceProp_t device_specs;

    HANDLE_ERROR(hipGetDeviceCount(&count));

    for (int i = 0; i < count; i++) {
        HANDLE_ERROR(hipGetDeviceProperties(&device_specs, i));
    }



    // allocate memory on the cuda device
    HANDLE_ERROR(hipMalloc((void **) &cuda_gpu_ptr, sizeof(int)));

    // perform calculation with CUDA on GPU
    add<<<1, 1>>>(2, 7, cuda_gpu_ptr);


    // Copy result from CUDA memory back to host memory
    HANDLE_ERROR(hipMemcpy(&host_value, cuda_gpu_ptr, sizeof(int), hipMemcpyDeviceToHost));
    // end utility macro

    printf("3+7 = %d\n", host_value);
    hipFree(cuda_gpu_ptr);

    return 0;
}
